#include "hip/hip_runtime.h"
#ifndef _FGAC_COMPRESS_TEXTURE_CU_
#define _FGAC_COMPRESS_TEXTURE_CU_

#include "fgac_internal.cuh"
#include "fgac_compress_texture.h"
#include "fgac_compress_symbolic.cuh"
#include "fgac_image.cuh"

// note:
// 1. don't support hdr compression for now
// 2. only support u8 format

__global__ void GPUEncodeKernel(uint8_t* outputData, hipTextureObject_t tex, fgac_contexti* ctx)
{
	// calculate normalized texture coordinates
	uint32_t global_index_x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t global_index_y = blockIdx.y * blockDim.y + threadIdx.y;
	
	const block_size_descriptor bsd = ctx->bsd;

	// block size e.g. 8x8
	int block_x = bsd.xdim;
	int block_y = bsd.ydim;

	// image size
	int dim_x = ctx->dim_x;
	int dim_y = ctx->dim_y;

	// the number of the blocks of each raw
	int xblocks = (dim_x + block_x - 1) / block_x;
	int yblocks = (dim_y + block_y - 1) / block_y;

	int offset = (( yblocks + global_index_y) * xblocks + global_index_x) * 16;

	if (global_index_x >= xblocks || global_index_y >= dim_y)
	{
		return;
	}

	uint2 start_pix_pos = make_uint2(global_index_x, global_index_y) * make_uint2(block_x, block_y);

	image_block blk;
	uint8_t* dstData = outputData + offset;
	compression_working_buffers tmpBuf;
	load_image_block_fast_ldr(&blk, start_pix_pos, tex, ctx);
	compress_block(ctx,&blk, dstData,&tmpBuf);
}

extern "C" void GPUEncodeKernel(dim3 gridSize, dim3 blockSize, uint8_t * outputData, hipTextureObject_t tex, fgac_contexti * ctx)
{
	GPUEncodeKernel << <gridSize, blockSize, 0 >> > (outputData,tex, ctx);
}

#endif